#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#define DIM  64
#define XDIM  DIM
#define YDIM  DIM
#define MATRIXSIZE  XDIM*YDIM
#define BLOCKSPERGRID  16
#define NUMTHREADS DIM

/*****************************************************************************/
__global__ void multMatrix(const int *A, const int *B, int *C, int numElements)
{
	int yOffset;
    int i, x;
    __shared__ int rowAshared[XDIM][32];

    int y = blockDim.x * blockIdx.x + threadIdx.x;
    int yRel = y - (blockDim.x * blockIdx.x);

    yOffset = y * XDIM;
    for(i = 0; i < XDIM; i++)
        rowAshared[i][yRel] = *(A + yOffset + i);

    if (y < numElements)
    {
        for(x = 0; x < XDIM; x++)
        {   *(C + yOffset + x) = 0;
            for(i = 0; i < XDIM; i++){
                *(C + yOffset + x) = *(C + yOffset + x) + (rowAshared[i][yRel] * (*(B + (i*YDIM) + x )));
            }
        } 
    }
}


/*****************************************************************************/

int printMatrix(int *ap)
{
	int x, y;
	for(y = 0; y < YDIM; y++)
	{
		printf("\n");
		for(x = 0; x < XDIM; x++)
		{
		    printf("%i ", *(ap + (y*XDIM) + x));
		}
	}
	printf("\n");
return 0;
}


/******************************************************************************
 * Host main routine
 */
int main(int argc, char *argv[])
{   
    if (argc != 3) {
        cout << "Error en numero de parametros de entrada" << endl;
        exit(0);
    }
    else {
        int N = stoi(argv[1]);
        int NUMTHREADS = stoi(argv[2]);
        int threadsPerBlock = (N + NUMTHREADS - 1) / NUMTHREADS;
        int XDIM = N;
        int YDIM = N;
        INT MATRIXSIZE = XDIM*YDIM;
    }
    int i, v=0;
    int blocksPerGrid, threadsPerBlock;
    blocksPerGrid = BLOCKSPERGRID;
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = MATRIXSIZE;
    size_t size = MATRIXSIZE * sizeof(int);
    if(v == 1) printf("[Matrix mult of %d elements]\n", numElements);

    // Allocate the host input vector A
    int *h_A = (int *)malloc(size);

    // Allocate the host input vector B
    int *h_B = (int *)malloc(size);

    // Allocate the host output vector C
    int *h_C = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    
    for(i = 0; i < MATRIXSIZE; i++){
        *(h_A + i) = rand() & 0xF;
        *(h_B + i) = rand() & 0xF;        
        *(h_C + i) = 0;
    }
    //printMatrix(h_A);
    //printMatrix(h_B);
    
    // Allocate the device input vector A
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    int *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    int *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    if(v == 1) printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    threadsPerBlock = NUMTHREADS/blocksPerGrid;
    //blocksPerGrid = BLOCKS; //(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    multMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printMatrix(h_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    return 0;
}
